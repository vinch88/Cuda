#include "hip/hip_runtime.h"

#include "Indice1D.h"
#include "cudaTools.h"
#include <stdio.h>
#include "reductionADD.h"
#include <hiprand/hiprand_kernel.h>

static __device__ void reductionInterThread(hiprandState* ptrTabGen, int nbFlecheByThread);
static __device__ float f(float x);

__global__ void montecarlo(hiprandState* ptrTabGen, int nbFlecheByThread, int* ptrDevNxTotal)
    {
    extern __shared__ int tab_SM[];
    reductionInterThread(ptrTabGen, nbFlecheByThread);
    __syncthreads();
    reductionADD<int>(tab_SM, ptrDevNxTotal);
    }

__device__
void reductionInterThread(hiprandState* ptrTabGen, int nbFlecheByThread)
    {
    const int TID_LOCAL = Indice1D::tidLocal();
    const int TID_GLOBAL = Indice1D::tid();
    const int NB_THREAD = Indice1D::nbThread();

    hiprandState localGenerator = ptrTabGen[TID_GLOBAL];

    float xAlea;
    float yAlea;

    float sum = 0;

    for(long i = 0; i < nbFlecheByThread; i++)
	{
	xAlea = hiprand_uniform(&localGenerator);
	yAlea = hiprand_uniform(&localGenerator);

	sum += f(i);

	}

    ptrTabGen[TID_LOCAL] = localGenerator;
    }

__device__
float f(float x)
    {
    return 4.0 / (1 + x * x);
}
