#include "hip/hip_runtime.h"
Raytracing ::Raytracing(int nbSphere, int w, int h, . . . )
    {
    . . .
    ShereCreator shereCreator(nbSphere,w,h); // sur la pile
    Sphere* ptrTabSphere=shereCreator.getTab();
// transfert to GM
    toGM(ptrTabSphere);// a implemneter
// transfert to CM
    toCM(ptrTabSphere);// a implemneter
    } // shereCreator qui est sur la pile est détruit ici
// ce qui détruit les sphères cotés host, via son destructeur

#include "Raytracing.h"

#include <iostream>
#include <assert.h>

#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void raytracing(uchar4* ptrDevPixels,uint w, uint h,float t);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Raytracing::Raytracing(const Grid& grid, uint w, uint h, float dt) :
	Animable_I<uchar4>(grid, w, h, "Raytracing_Cuda_RGBA_uchar4")
    {
    // Inputs
    this->dt = dt;

    // Tools
    this->t = 0; // protected dans Animable
    }

Raytracing::~Raytracing()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void Raytracing::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("raytracing rgba uchar4 (before)"); // facultatif, for debug only, remove for release

    raytracing<<<dg,db>>>(ptrDevPixels,w,h,t);

    Device::lastCudaError("raytracing rgba uchar4 (after)"); // facultatif, for debug only, remove for release
    }

/**
 * Override
 * Call periodicly by the API
 */
void Raytracing::animationStep()
    {
    t += dt;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

