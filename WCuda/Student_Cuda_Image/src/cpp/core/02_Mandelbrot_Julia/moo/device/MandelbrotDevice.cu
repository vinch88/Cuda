#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "math/MandelbrotMath.h"
#include "DomaineMath_GPU.h"
#include "IndiceTools_GPU.h"
using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/
__global__ void mandelbrot(uchar4* ptrDevPixels, uint w, uint h, uint t, DomaineMath domaineMath)
    {
    MandelbrotMath mandelbrotMath = MandelbrotMath(t);

    const int WH=w*h;
    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();

    int i = 0;
    int j = 0;

    double x = 0;
    double y = 0;

    int s = TID;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &i, &j);

	domaineMath.toXY(i, j, &x, &y);

	mandelbrotMath.colorXY(&ptrDevPixels[s],x, y);
	s += NB_THREAD;
	}
    }


/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

